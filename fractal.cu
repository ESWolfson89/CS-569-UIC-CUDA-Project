#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include "GL/glut.h"
#include <math.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

using namespace std;

static const int WID = 768;
static const int HGT = 768;
static const unsigned MaxIterations = 500;

clock_t clock_timer_var;

#define BLOCK_SIZE 16

int main_window;

// not used
struct color
{
	GLdouble r, g, b;
};

__global__ void drawMandelbrotGPU(double MaxRe, double MinRe, double MaxIm, double MinIm, color *cell_array);

double cell_arrayr[WID * HGT];

int iteration = 1;
bool GPU_mode = false;
bool update_screen = true;
double MinRe = -2.0;
double MaxRe = 1.0;
double MinIm = -1.5;
double MaxIm = 1.5;
double by = 1.0;
double bx = 1.0;
double ey = 1.0;
double ex = 1.0;
double cy = 1.0, cx = 1.0;
bool draw = true;

void resetZoomParameters()
{
   MinRe	=	-2.0;
   MaxRe	=	1.0;
   MinIm	=	-1.5;
   MaxIm	=	1.5;
}

void init(int *argc_ptr, char** argv)
{
   glClearColor(0.0,0.0,0.0,0.0);       // background color is black
   glColor3f(0.0f, 0.0f, 0.0f);         // drawing color is black (initially) 
   glPointSize(1.0);                          // a 'dot' is 1 by 1 pixel
   glMatrixMode(GL_PROJECTION);            // set "camera shape"
   glLoadIdentity();
   gluOrtho2D(0.0, (GLdouble)WID, 0.0, (GLdouble)HGT);
}

void drawSavedMap()
{
	glBegin(GL_POINTS);
	color c;
	for (int i = 0; i < WID; ++i)
	{
		for (int j = 0; j < HGT; ++j)
		{
			c.r = cell_arrayr[i*WID + j];	
			glColor3f(c.r,c.r/4,0.0f);
			glVertex2f(i,j);
		}
	}
	glEnd();
}	

void printInfo()
{
    cout << "Last redraw/compute:" << endl;
    cout << "size of double: " << sizeof(double) << " size of long double: " << sizeof(long double) << endl;

    if (GPU_mode == false)
        cout << "********CPU MODE********";
    else
        cout << "********GPU MODE********";

    cout << endl << "Max iterations: " << MaxIterations << " BLOCK_SIZE: " << BLOCK_SIZE;

    cout << endl << "Min x (real axis): " << MinRe << endl << "Max x (real axis): " << MaxRe;
    cout << endl << "Min y (complex axis): " << MinIm << endl << "Max y (complex axis): " << MaxIm;
    cout << endl << "Time to calculate Mandelbrot & draw screen: " << (clock() - clock_timer_var ) / (double) CLOCKS_PER_SEC << "s";
    cout << endl;
}

void drawBox()
{
    glColor3f(1.0f,1.0f,1.0f);
    glBegin(GL_LINES);
    glVertex2f(bx,HGT-by);
    glVertex2f(cx,HGT-by);
    glEnd();
    glBegin(GL_LINES);
    glVertex2f(bx,HGT-by);
    glVertex2f(bx,cy);
    glEnd();
    glBegin(GL_LINES);
    glVertex2f(cx,HGT-by);
    glVertex2f(cx,cy);
    glEnd();
    glBegin(GL_LINES);
    glVertex2f(bx,cy);
    glVertex2f(cx,cy);
    glEnd();
}

__global__ void drawMandelbrotGPU(double MaxRe, double MinRe, double MaxIm, double MinIm, double *cell_arrayr)
{
  // r stands for row, c for column
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int c = blockIdx.x * blockDim.x + threadIdx.x; 
  int index = (c * HGT) + r;

  if(c >= WID || r >= HGT) 
	 return;

  double Re_factor	= (MaxRe - MinRe) / WID;
  double Im_factor	= (MaxIm - MinIm) / HGT;  
  double c_im, c_re, Z_re, Z_im, Z_re2, Z_im2;
  double loop_iters_n;

  c_im = MaxIm - r * Im_factor;
  c_re = MinRe + c * Re_factor;
  Z_re = c_re;
  Z_im = c_im;
  loop_iters_n = 0;

  for (unsigned n = 0; n < MaxIterations; ++n ) 
  {
      Z_re2 = Z_re * Z_re;
      Z_im2 = Z_im * Z_im;
      if ( Z_re2 + Z_im2 > 4 )
          break;
      Z_im = 2 * Z_re * Z_im + c_im;
      Z_re = Z_re2 - Z_im2 + c_re;
      loop_iters_n = (double)n;
  }

  loop_iters_n /= MaxIterations;
  cell_arrayr[index] = loop_iters_n;
}

void drawMandelbrotCPU()
{
  glClear(GL_COLOR_BUFFER_BIT);
  double Re_factor	=	(MaxRe - MinRe) / WID;
  double Im_factor	=	(MaxIm - MinIm) / HGT;  
  double c_im, c_re, Z_re, Z_im, Z_re2, Z_im2;
  double loop_iters_n;
  
  glBegin(GL_POINTS);

  for ( unsigned y = 0; y < HGT; ++y ) 
  {
	  c_im = MaxIm - y * Im_factor;
	  for ( unsigned x = 0; x < WID; ++x ) 
	  {
		  c_re = MinRe + x * Re_factor;
		  Z_re = c_re;
		  Z_im = c_im;
		  color col;
		  loop_iters_n = 0;
		  for ( unsigned n = 0; n < MaxIterations; ++n ) 
		  {	
			  Z_re2 = Z_re * Z_re;
			  Z_im2 = Z_im * Z_im;
			  if ( Z_re2 + Z_im2 > 4 )
				  break;
			  Z_im = 2 * Z_re * Z_im + c_im;
			  Z_re = Z_re2 - Z_im2 + c_re;
			  loop_iters_n = (double)n;
		  }
		  loop_iters_n /= MaxIterations;
		  col.r = loop_iters_n;
		  glColor3f(col.r,col.r/4,0.0f);
		  glVertex2f(x,y);
		  cell_arrayr[(x*WID)+y] = loop_iters_n;
	  }
  }

  glEnd();
}

void reDisplay(void)
{
  if (update_screen)
  {
  	if (draw == true)
  	{
	  clock_timer_var = clock();

	  if (GPU_mode == false)
  		drawMandelbrotCPU();
	  else
	  {
		double *array_temp;
		size_t size = sizeof(double) * WID * HGT;
		hipMalloc(&array_temp,size);
		hipMemcpy(array_temp,cell_arrayr,size,hipMemcpyHostToDevice);
		dim3 b_size(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(WID / b_size.x, HGT / b_size.y);	
		drawMandelbrotGPU<<<grid,b_size>>>(MaxRe, MinRe, MaxIm, MinIm, array_temp);
		hipDeviceSynchronize();
		hipMemcpy(cell_arrayr,array_temp,size,hipMemcpyDeviceToHost);
		hipFree(array_temp);
		glClear(GL_COLOR_BUFFER_BIT);
		drawSavedMap();
	  }
          printInfo();
          update_screen = false;
  	}
  	else
  	{
  		drawSavedMap();
  		drawBox();
  	}
  	glFlush();
  }	
}


void keyPress(unsigned char k, int x, int y)
{
	if (k == 'g')
		GPU_mode = true;
	if (k == 'c')
		GPU_mode = false;
	if (k == 'q')
		exit(0);
	if (k == '1')
	{
		MinRe = -0.480469;
		MaxRe = 0.179688;
		MinIm = -0.402344;
		MaxIm = 0.371094;
		draw = true;
		update_screen = true;
	}
	if (k == '2')
	{
		MinRe = -1.51172;
		MaxRe = -0.820313;
		MinIm = -1.18359;
		MaxIm = -0.675781;
		draw = true;
		update_screen = true;
	}
	if (k == '3')
	{
		MinRe = -0.806519;
		MaxRe = -0.776896;
		MinIm = -0.168213;
		MaxIm = -0.137777;
		draw = true;
		update_screen = true;
	}
	if (k == '4')
	{
		MinRe = -0.755904;
		MaxRe = -0.75399;
		MinIm = -0.0578789;
		MaxIm = -0.0563382;
		draw = true;
		update_screen = true;
	}
	if (k == '5')
	{
		MinRe = 0.269364;
		MaxRe = 0.269776;
		MinIm = -0.00462961;
		MaxIm = -0.00434589;
		draw = true;
		update_screen = true;
	}
	if (k == 'r')
	{
            resetZoomParameters();
            draw = true;		
            update_screen = true;
	}
}

void mouseMove(int x, int y)
{
	cx = x;
	cy = y;
  update_screen = true;
}

void mouseClick(int button, int state, int x, int y)
{
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		draw = false;
		by = cy = y;
		bx = cx = x;
	}
	else if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
	{
		draw = true;
		ey = y;
		ex = x;
		double tempMinRe = MinRe, tempMaxRe = MaxRe;
		double tempMinIm = MinIm, tempMaxIm = MaxIm;
		MinRe = ((min(bx,ex)/WID) * (tempMaxRe - tempMinRe)) + tempMinRe;
		MaxRe = ((max(bx,ex)/WID) * (tempMaxRe - tempMinRe)) + tempMinRe;
		MinIm = ((min(by,ey)/HGT) * (tempMaxIm - tempMinIm)) + tempMinIm;
		MaxIm = ((max(by,ey)/HGT) * (tempMaxIm - tempMinIm)) + tempMinIm;
		update_screen = true;
	}
}

void wrapDisplay(void) { reDisplay(); }

void wrapMouse(int b, int st, int x, int y)
{
	mouseClick(b,st,x,y);
}

void wrapMotion(int x, int y)
{
	mouseMove(x,HGT-y);
}

void wrapKey(unsigned char k, int x, int y)
{
	keyPress(k,x,(HGT-y));
}

void timer_func(int value)     
{     
	glutPostRedisplay();
	glutTimerFunc(1, timer_func, 0);
}


//<<<<<<<<<<<<<<<<<<<<<<<< main >>>>>>>>>>>>>>>>>>>>>>
int main(int argc, char* argv[])
{
      srand(time(NULL));
      
      glutInit(&argc, argv);        
      glutInitWindowSize(WID, HGT);
      glutInitWindowPosition(100, 100); 
      glutInitDisplayMode(GLUT_RGB); 
      
      main_window = glutCreateWindow( "Mandelbrot By Eric Wolfson");
      
      init(&argc, argv); // initialize the display    
      
      glutDisplayFunc(wrapDisplay);   // register Redraw function
      glutMotionFunc(wrapMotion);
      glutMouseFunc(wrapMouse);
      glutKeyboardFunc(wrapKey);
      glutTimerFunc(1,timer_func, 0);
    
      glutMainLoop(); // perpetual loop
      hipDeviceReset();
      
      return 0;
} 
